#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossMaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  Forward_cpu(bottom, top);
}

template <typename Dtype>
void EuclideanLossMaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(EuclideanLossMaskLayer);

}  // namespace caffe
