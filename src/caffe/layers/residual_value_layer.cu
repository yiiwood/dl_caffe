#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ResidualForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* bottom_label, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int predt = static_cast<int>(bottom_data[index]);
    int label = static_cast<int>(bottom_label[index]);
    top_data[index] = label - predt;
  }
}

template <typename Dtype>
void ResidualValueLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_label = bottom[1]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();

  int count = bottom[0]->count();
  ResidualForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom_label, top_data);
}

template <typename Dtype>
void ResidualValueLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  // TODO(Yangqing): implement the GPU version of softmax.
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(ResidualValueLayer);

}  // namespace caffe
